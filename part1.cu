#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "file_io.c"

#define BLOCK_SIZE 32
#define TILE_WIDTH 2 

#define cudaCheckError() {                               \
  hipError_t e = hipGetLastError();                    \
  if (e != hipSuccess) {                                \
    printf("CUDA error %s:%d: %s\n", __FILE__, __LINE__, \
            hipGetErrorString(e));                      \
    exit(1);                                             \
  }                                                      \
}

// Initialize a vector of size m to 1
__global__ void ones(double* vec, int m)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if(row >= m) return;    
	vec[row] = 1.0;
}

// Calculate the value of w = average value of each row
__global__ void w_calc(double* objects, int m, int n, double *w)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if (row >= n) return;
    
   	double w_value = 0;
   	for (int i = 0; i < m ; i++)
   		w_value += objects[row * m + i];   
   	w[row] = w_value/m;
}

// Calculate tmp = A * xk = ( M - w*e' )*xk
__global__ void pddp1(double* objects, int m, int n, double *w,double *x,double *tmp)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if (row >= n ) return;

	double Cvalue = 0;
	
	for(int i = 0; i < m; i++)
		Cvalue += (objects[row * m + i] - w[row]) * x [i];
	tmp[row] = Cvalue;

	__syncthreads();
}

// Calculate output = A' * tmp = (M - w*e')' * ( M - w*e' )*xk
__global__ void pddp2(double* objects, double *output, int m, int n, double *w, double *tmp)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if (row >= m ) return;

	double final_value = 0;

	for(int j=0; j < n; j++)
		final_value += (objects[j*m+row] - w[j]) * tmp[j];
	output[row] = final_value; 
	
	__syncthreads();
}

// Calculate the power of each vector's vaule 
__global__ void power(double *input, int m , double *output)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;	
	if(row >= m ) return;

	output[row] = (double)powf(input[row] , 2);

	__syncthreads();
}

// Divide each element of a vector with a value
__global__ void division(double *input, int m, double norm, double *output)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if(row >= m ) return;

	output[row] = input[row] / norm;

	__syncthreads();
}

// Calculate the difference in order to diverge
__global__ void diff_pow(double *x, int m, double *y)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if(row >= m ) return;

	x[row] = y[row] - x[row];
	x[row] = (double)powf(x[row] , 2);
}

// Copy contents of vector c to vector x
__global__ void swap(double* x, int m, double *c)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	if(row >= m) return;   
	x[row] = c[row];
}

void StartKernelTiming (hipEvent_t& tic, hipEvent_t& toc, hipStream_t iStream)
{
	hipEventCreate(&tic);
	hipEventCreate(&toc);
	hipEventRecord(tic , iStream);
}

void StopKernelTiming (hipEvent_t& tic, hipEvent_t& toc, hipStream_t iStream, float* ptimer)
{
	float kt = 0;
	hipEventRecord(toc , iStream);
	hipEventSynchronize(toc);
	hipEventElapsedTime(&kt , tic , toc);
	hipEventDestroy(tic); hipEventDestroy(toc);
	(*ptimer) += kt;
}


int main(int argc, char **argv) {

	hipSetDevice(0);

	int n,m;
	char *input_file = argv[1];
	double *objects; 
	objects = file_read(input_file, &n, &m);
	printf("::Objects loaded::\n");
	printf("Objects: %d\n", m);
	printf("Attributes: %d\n", n);
 
	double fnorm, f_sum, final[m], eps;
	eps = pow(10,-6);
	double *w, *x, *tmp, *den, *y;
	w = (double*) malloc(n*sizeof(double));
	x = (double*) malloc(m*sizeof(double));
	tmp = (double*) malloc(n*sizeof(double));	
	den = (double*) malloc(m*sizeof(double));
	y = (double*)malloc(m*sizeof(double));	

	double *objects_d,*final_d,*w_d, *x_d,*tmp_d, *in_d, *den_d, *y_d;

  	dim3 dimBlock(1,BLOCK_SIZE);
    dim3 dimGrid(1,	(m + dimBlock.y - 1) / dimBlock.y);

    printf("Grid Size: (%d,%d) \n",dimGrid.x,dimGrid.y);
    printf("Block Size: (%d,%d) \n",dimBlock.x,dimBlock.y);
    
	hipMalloc((void**) &in_d , m*sizeof(double));
	hipMemcpy(in_d , x , m*sizeof(double) , hipMemcpyHostToDevice);
  	
  	hipMalloc((void**) &x_d , m*sizeof(double));
	hipMemcpy(x_d , x , m*sizeof(double) , hipMemcpyHostToDevice);
		
	hipMalloc((void **) &objects_d , m*n*sizeof(double));
	hipMemcpy (objects_d , objects , m*n*sizeof(double) , hipMemcpyHostToDevice );

	hipMalloc((void **) &final_d,m*sizeof(double));
	hipMalloc((void **) &tmp_d,n*sizeof(double));
	hipMalloc((void **) &w_d,n*sizeof(double));
	hipMalloc((void **) &den_d,m*sizeof(double));
	hipMalloc((void **) &y_d,m*sizeof(double));
	cudaCheckError();

	hipEvent_t tic, toc;
	float Elapsed_Time;

	StartKernelTiming(tic, toc, 0);
	ones<<<dimGrid,dimBlock>>>(x_d,m);
	StopKernelTiming(tic, toc, 0, &Elapsed_Time);
	hipMemcpy (x, x_d, m*sizeof(double), hipMemcpyDeviceToHost );
	cudaCheckError();

	StartKernelTiming(tic, toc, 0);
	w_calc<<<dimGrid,dimBlock>>>(objects_d,m,n,w_d);
	StopKernelTiming(tic, toc, 0, &Elapsed_Time);
	hipMemcpy (w , w_d , n*sizeof(double) , hipMemcpyDeviceToHost );

    do{
		StartKernelTiming(tic, toc, 0);
		pddp1<<<dimGrid,dimBlock>>>(objects_d,m,n,w_d,x_d,tmp_d);
		StopKernelTiming(tic, toc, 0, &Elapsed_Time);
		hipMemcpy (tmp , tmp_d , n*sizeof(double) , hipMemcpyDeviceToHost );

		StartKernelTiming(tic, toc, 0);
		pddp2<<<dimGrid,dimBlock>>>(objects_d,final_d,m,n,w_d,tmp_d);
		StopKernelTiming(tic, toc, 0, &Elapsed_Time);
		hipMemcpy (final , final_d , m*sizeof(double) , hipMemcpyDeviceToHost );
		
		//calculate each elements square
		StartKernelTiming(tic, toc, 0);
		power<<<dimGrid,dimBlock>>>(final_d,m,den_d);
		StopKernelTiming(tic, toc, 0, &Elapsed_Time);
		hipMemcpy (den , den_d , m*sizeof(double) , hipMemcpyDeviceToHost);
		cudaCheckError();

		//sum all elements
		StartKernelTiming(tic, toc, 0);
		f_sum = 0.0;
		for (int i = 0 ; i<m; i++)
			f_sum+=den[i];
		f_sum = sqrt(f_sum);
		StopKernelTiming(tic, toc, 0, &Elapsed_Time);

		//divide each element of y with norm
		StartKernelTiming(tic, toc, 0);
		division<<<dimGrid,dimBlock>>>(final_d,m,f_sum,y_d);
		StopKernelTiming(tic, toc, 0, &Elapsed_Time);
		hipMemcpy (y, y_d,m* sizeof(double), hipMemcpyDeviceToHost );
		cudaCheckError();

		//calculate difference xk+1 - xk and find each element's square
		StartKernelTiming(tic, toc, 0);
		diff_pow<<<dimGrid,dimBlock>>>(x_d,m,y_d);
		StopKernelTiming(tic, toc, 0, &Elapsed_Time);
		hipMemcpy (x, x_d, m*sizeof(double), hipMemcpyDeviceToHost );
		cudaCheckError();

		//calculate final norm 
		StartKernelTiming(tic, toc, 0); 
		fnorm = 0.0;
		for (int i = 0 ; i<m; i++)
			fnorm+=x[i];
		fnorm = sqrt(fnorm);
		StopKernelTiming(tic, toc, 0, &Elapsed_Time);

		StartKernelTiming(tic, toc, 0);
		swap<<<dimGrid,dimBlock>>>(in_d,m,y_d);
		StopKernelTiming(tic, toc, 0, &Elapsed_Time);
		hipMemcpy (x, in_d, m*sizeof(double), hipMemcpyDeviceToHost );
		hipMemcpy(x_d,x,m*sizeof(double),hipMemcpyHostToDevice);
		cudaCheckError();

	}while(fnorm > eps);
	
	printf("-----------------\n");
	printf("Elapsed_Time=%f ms\n", Elapsed_Time);

	//Print the last 5 values of y to show correctness
	printf("-----------------\n");
	for(int i=m-5; i<m; i++)
		printf("y[%d] = %.7f \n",i,y[i]);

	hipFree(objects_d);
	hipFree(final_d);
	hipFree(w_d);
	hipFree(x_d);
	hipFree(tmp_d);
	hipFree(in_d);
	hipFree(den_d);
	hipFree(y_d);
	free(w);
	free(x);
	free(tmp);
	free(den);
	free(y);
	free(objects);

	return (0);
}

